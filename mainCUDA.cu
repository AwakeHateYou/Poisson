//
// Poisson 
//

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <string.h>
#include <hipfft/hipfft.h>


#define PI 3.141592
#define delta 0.1
using namespace std;

#define BLOCK_SIZE 	16			// submatrix size
#define	N			128		// matrix size is N*N

double S(double  x, double y){
	return ((x - 1.0 / 2.0)*(x - 1.0 / 2.0) + (y - 1.0 / 2.0)*(y - 1.0 / 2.0));
}
void initU(double *mass, double dx){
	for (int j = 0; j < N; j++)
	for (int i = 0; i < N; i++)
		mass[i + N*j] = ((S(dx*i, dx*j) - 2.0 * delta*delta) / (delta*delta*delta*delta))
		*exp(-S(dx*i, dx*j) / 2.0*delta*delta);
}
void printGNU(double *mass, double dx, char* filename){
	ofstream fout;
	fout.open(filename);
	//fout.precision(3);
	for (int j = 0; j < N; j++){
		for (int i = 0; i < N; i++)
			fout << dx*i << ' ' << dx*j << ' ' << mass[i + j*N] << endl;
		fout << endl;
	}
	fout.close();
}

// KERNEL //
__global__ void matMult(hipfftDoubleReal *Ug, hipfftDoubleComplex *complex, int n){
	int   bx = blockIdx.x;		// block index
	int   by = blockIdx.y;
	int   tx = threadIdx.x;		// thread index
	int   ty = threadIdx.y;		

	int   idx = BLOCK_SIZE * bx + tx;
	int	  idy = BLOCK_SIZE * by + n * ty;


	
}


// HOST CODE //
int main(int argc, char *  argv[]){
	double dx = 1.0 / N;
	int	numBytesD = N * N * sizeof (hipfftDoubleReal);
	int numBytesC = N * (N / 2 + 1) * sizeof (hipfftDoubleComplex);

	// allocate host memory
	double *U = new double[N*N];
	initU(U, dx);

	// allocate device memory
	hipfftDoubleReal *Ug;
	hipfftDoubleComplex  *complex;
	hipfftHandle ahead, backward;

	//cudaMalloc((void**)&ahead, sizeof(cufftHandle));
	//cudaMalloc((void**)&backward, sizeof(cufftHandle));
	hipMalloc((void**)&complex, numBytesC);
	hipMalloc((void**)&Ug, numBytesD);

	hipfftPlan2d(&ahead, N, N, HIPFFT_D2Z); 
	hipfftPlan2d(&backward, N, N, HIPFFT_Z2D);



	// set kernel launch configuration
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(N / threads.x, N / threads.y);

	// create cuda event handles
	hipEvent_t start, stop;
	float gpuTime = 0.0f;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// asynchronously issue work to the GPU (all to stream 0)
	hipEventRecord(start, 0);
	hipMemcpy(Ug, U, numBytesD, hipMemcpyHostToDevice);

	hipfftExecD2Z(ahead, Ug, complex);
	matMult <<<blocks, threads >>> (Ug, complex, N);
	hipfftExecZ2D(backward, complex, Ug);
	

	hipMemcpy(U, Ug, numBytesD, hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);

	// print the cpu and gpu times
	printf("time spent executing by the GPU: %.2f millseconds\n", gpuTime);

	// release resources
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(Ug);
	hipFree(complex);
	hipfftDestroy(ahead);
	hipfftDestroy(backward);

	delete[]U;


	return 0;
}
